#include "hip/hip_runtime.h"
// SPDX-License-Identifier: Apache-2.0
//
// quadriga-lib c++/MEX Utility library for radio channel modelling and simulations
// Copyright (C) 2022-2024 Stephan Jaeckel (https://sjc-wireless.com)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

#include "quadriga_CUDA_tools.cuh"

// KERNEL: Add two numbers
__global__ void Add_A_and_B(float *d_a, float *d_b, float *d_c)
{
    d_c[0] = d_a[0] + d_b[0];
}

// Returns the compute capability of the NVIDIA GPU
double quadriga_lib::get_CUDA_compute_capability(int device)
{
    // Initialize CUDA Error
    hipError_t error;

    // Connect which GPU to run on
    error = hipSetDevice(device);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    // Host variables
    float h_a = 3.0, h_b = 7.0;

    // Pointer to device variables
    float *d_a, *d_b, *d_c;
    size_t sz = sizeof(float);

    // Allocate memory on device
    error = hipMalloc(&d_a, sz);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    error = hipMalloc(&d_b, sz);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    error = hipMalloc(&d_c, sz);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    // Copy data to device
    error = hipMemcpy(d_a, &h_a, sz, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    error = hipMemcpy(d_b, &h_b, sz, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    // Perform simple computation on the device
    Add_A_and_B<<<1, 1>>>(d_a, d_b, d_c);

    // Check for error
    error = hipPeekAtLastError();
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    // Retrieve data from device
    float *h_c = new float[1];
    error = hipMemcpy(h_c, d_c, sz, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }
    if (h_c[0] != 10)
    {
        hipDeviceReset();
        return 0.0;
    }
    delete[] h_c;

    int *val = new int[1];
    error = hipDeviceGetAttribute(val, hipDeviceAttributeComputeCapabilityMinor, 0);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    double cc = (double)val[0];
    error = hipDeviceGetAttribute(val, hipDeviceAttributeComputeCapabilityMajor, 0);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return 0.0;
    }

    cc = cc * 0.1 + (double)val[0];
    delete[] val;

    hipDeviceReset();
    return cc;
}
